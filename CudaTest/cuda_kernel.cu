#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "./cuda_kernel.cuh"

__global__ void vectorAdditionKernel(double* A, double* B, double* C, int arraySize) {
	int threadID = blockDim.x * blockIdx.x + threadIdx.x;

	if (threadID < arraySize) {
		C[threadID] = A[threadID] + B[threadID];
	}
}



void kernel(double* A, double* B, double* C, int arraySize) {
	double* d_A, * d_B, * d_C;

	hipMalloc((void**)&d_A, arraySize * sizeof(double));
	hipMalloc((void**)&d_B, arraySize * sizeof(double));
	hipMalloc((void**)&d_C, arraySize * sizeof(double));

	hipMemcpy(d_A, A, arraySize * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, arraySize * sizeof(double), hipMemcpyHostToDevice);

	dim3 blockSize(512, 1, 1);
	dim3 gridSize(512 / arraySize + 1, 1);

	vectorAdditionKernel << <gridSize, blockSize>> >(d_A, d_B, d_C, arraySize);

	hipMemcpy(C, d_C, arraySize * sizeof(double), hipMemcpyDeviceToHost);
}